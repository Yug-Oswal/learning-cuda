#include <stdio.h>
#include <hip/hip_runtime.h>

// __global__ indicates this function runs on the GPU.
__global__ void helloFromGPU() {
    if (threadIdx.x == 0) {
        printf("Hello from the GPU!\n");
    }
}

int main() {
    // Each block output printed and thread output prolly overlap
    helloFromGPU<<<2, 10>>>();

    // Synchronize to ensure the GPU finishes before exiting.
    hipDeviceSynchronize();

    printf("Hello from the CPU!\n");
    return 0;
}